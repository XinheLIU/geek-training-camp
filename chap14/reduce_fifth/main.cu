
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <chrono>

using namespace std::chrono;

__device__ void warpReduce(volatile float *sdata, int tid) {
    sdata[tid] += sdata[tid + 32];
    sdata[tid] += sdata[tid + 16];
    sdata[tid] += sdata[tid + 8];
    sdata[tid] += sdata[tid + 4];
    sdata[tid] += sdata[tid + 2];
    sdata[tid] += sdata[tid + 1];
}

__global__ void reduce5(float *g_idata, float *g_odata) {
    extern __shared__ float sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    sdata[tid] = g_idata[i] + g_idata[i + blockDim.x];
    __syncthreads();
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) { sdata[tid] += sdata[tid + s]; }
        __syncthreads();
    }

    if (tid > 32) warpReduce(sdata, tid);
}

int main(void) {
    int N = 100000000;
    float *g_indata_host, *g_indata_device, *g_outdata_host, *g_outdata_device;
    g_indata_host = (float *) malloc(N * sizeof(float));
    g_outdata_host = (float *) malloc(sizeof(float));

    hipMalloc(&g_indata_device, N * sizeof(float));
    hipMalloc(&g_outdata_device, sizeof(float));

    for (auto i = 0; i < N; i++) {
        g_indata_host[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);;
    }

    hipMemcpy(g_indata_device, g_indata_host, N * sizeof(float), hipMemcpyHostToDevice);

//    This is where the code is run
    auto start = high_resolution_clock::now();
    reduce5<<<(N + 255) / 256, 256>>>(g_indata_device, g_outdata_device);
    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    std::cout << "Time taken by function: "
              << duration.count() << " microseconds" << std::endl;
    hipFree(g_indata_device);
    hipFree(g_outdata_device);
    free(g_indata_host);
    free(g_outdata_host);

}